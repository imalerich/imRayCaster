#include "hip/hip_runtime.h"
#define GLEW_STATIC
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>

#include "gl_util.hpp"

const char * WINDOW_TITLE = "RayCaster - Cuda";
void present_gl();

surface<void, 2> tex;
__global__ void runCuda(float4 * tex, int screen_w, int screen_h) {
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < screen_w && y < screen_h) {
		float val = x / (float)screen_w;
		tex[y + screen_w + y] = make_float4(val, 1.0f, 0.0f, 1.0f);
	}
}

void check_err(hipError_t err) {
	if (err != hipSuccess) {
		printf("%s\n", hipGetErrorString(err));
	} else {
		printf("CUDA returned success.\n");
	}
}

int main() {
	init_gl(WINDOW_TITLE, VSYNC_ENABLED);

	struct hipGraphicsResource * tex_res;
	hipGraphicsGLRegisterImage(&tex_res, screen_tex, GL_TEXTURE_2D, cudaGraphicsMapFlagsWriteDiscard);
	float4 * d_tex;
	hipGraphicsMapResources(1, &tex_res, 0);
	size_t num_bytes;
	check_err(hipGraphicsResourceGetMappedPointer((void **)&d_tex, &num_bytes, tex_res));

	dim3 block(16, 16);
	dim3 grid((screen_w + block.x - 1) / block.x,
			  (screen_h + block.y - 1) / block.y);
	runCuda<<<grid, block>>>(d_tex, screen_w, screen_h);
	hipGraphicsUnmapResources(1, &tex_res, 0);

	// Game loop.
	glfwSetTime(0.0f);
	while (!glfwWindowShouldClose(window)) {
		// Close on escape press.
		if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS) {
			glfwSetWindowShouldClose(window, GL_TRUE);
		}

		// TODO

		present_gl();
		glfwSwapBuffers(window);
		glfwPollEvents();
	}

	// Done - cleanup
	glfwTerminate();
	return 0;
}

/**
 * Push a new frame to the screen.
 * This will contain the 'screen_tex' managed by gl_util.
 */
void present_gl() {
	glClearColor(0, 0, 0, 1);
	glClear(GL_COLOR_BUFFER_BIT);
	update_screen();
}
