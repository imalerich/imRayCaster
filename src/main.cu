#include "hip/hip_runtime.h"
#define GLEW_STATIC
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>

#include "gl_util.hpp"

const char * WINDOW_TITLE = "RayCaster - Cuda";
void present_gl();

surface<void, cudaSurfaceType2D> tex;
__global__ void runCuda(int screen_w, int screen_h) {
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < screen_w && y < screen_h) {
		float val = x / (float)screen_w;
		float4 data = make_float4(val, val, val, 1.0f);
		surf2Dwrite<float4>(data, tex, x * sizeof(float4), y);
	}
}

void check_err(hipError_t err) {
	if (err != hipSuccess) {
		printf("%s\n", hipGetErrorString(err));
		exit(0);
	} else {
		printf("CUDA returned success.\n");
	}
}

int main() {
	init_gl(WINDOW_TITLE, VSYNC_ENABLED);

	struct hipGraphicsResource * tex_res;
	struct hipArray * cu_arr;

	hipSetDevice(0);
	cudaGLSetGLDevice(0);
	hipGraphicsGLRegisterImage(&tex_res, screen_tex, GL_TEXTURE_2D, cudaGraphicsMapFlagsWriteDiscard);
	hipGraphicsMapResources(1, &tex_res, 0);
	hipGraphicsSubResourceGetMappedArray(&cu_arr, tex_res, 0, 0);
	hipError_t err = cudaBindSurfaceToArray(tex, cu_arr);
	check_err(err);

	dim3 block(16, 16);
	dim3 grid((screen_w + block.x - 1) / block.x,
	 		  (screen_h + block.y - 1) / block.y);
	runCuda<<<grid, block>>>(screen_w, screen_h);
	hipGraphicsUnmapResources(1, &tex_res, 0);
	hipStreamSynchronize(0);

	// Game loop.
	glfwSetTime(0.0f);
	while (!glfwWindowShouldClose(window)) {
		// Close on escape press.
		if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS) {
			glfwSetWindowShouldClose(window, GL_TRUE);
		}

		// TODO

		present_gl();
		glfwSwapBuffers(window);
		glfwPollEvents();
	}

	// Done - cleanup
	glfwTerminate();
	return 0;
}

/**
 * Push a new frame to the screen.
 * This will contain the 'screen_tex' managed by gl_util.
 */
void present_gl() {
	glClearColor(0, 0, 0, 1);
	glClear(GL_COLOR_BUFFER_BIT);
	update_screen();
}
